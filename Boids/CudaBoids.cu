#include "hip/hip_runtime.h"
#include <math.h>
#include <assert.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <cuda_gl_interop.h>
//#include <cudpp.h> //Cuda Parallel Primitives

//static CUDPPHandle* theCudpp;

extern "C" {
	#include "util.h"
}

union BoidPack {
	char buffer[sizeof(Boid)];
	Boid m_Boid;
};

#define check_cuda_error() {								\
	if (hipError_t e = hipGetLastError()) {				\
		fprintf(stderr, "%s:%i: %s\n", __FILE__, __LINE__,	\
				hipGetErrorString(e));						\
		exit(-1);											\
	}														\
}

__global__ static void neighbourhood(int *neighbours, unsigned int *flags,
		const float *distances, const int n, const int EPSILON) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x,
		iy = blockIdx.y * blockDim.y + threadIdx.y,
		offset = ix + iy * n;
	if (ix < n && iy < n && distances[offset] <= EPSILON) {
		neighbours[offset] = ix;
		flags[offset] = 1;
	}
}
/*
__global__ static void compact(int *neighbours,
		const unsigned int *scanned_flags, const int n) {
	const int delim = INT_MAX;
	int ix = blockIdx.x * blockDim.x + threadIdx.x,
		iy = blockIdx.y * blockDim.y + threadIdx.y,
		offset = iy * n;
//#define scanned_flags(x) tex1Dfetch(flags_texture, (x))
	if (iy < n) {
		if (0 == ix) {
			neighbours[offset + scanned_flags(offset + n - 1)] = delim;
		} else if (ix < n && scanned_flags(offset + ix) >
				scanned_flags(offset + ix - 1)) {
			neighbours[offset + scanned_flags(offset + ix) - 1] =
				neighbours[offset + ix];
		}
	}
#undef scanned_flags
}

static CUDPPHandle prepare_scan_plan(int n, size_t pitch) {
	CUDPPResult r = cudppCreate(theCudpp);
		if (CUDPP_SUCCESS != r) {
		printf("Error creating CUDPPHandle\n");
		exit(-1);
	}
	CUDPPConfiguration config;
	config.datatype = CUDPP_UINT;
	config.algorithm = CUDPP_SCAN;
	config.options = CUDPP_OPTION_FORWARD;
	config.op = CUDPP_ADD;
	CUDPPHandle planhandle = 0;
	CUDPPResult result = cudppPlan(*theCudpp, &planhandle, config, n, n, pitch);
	if (CUDPP_SUCCESS != result) {
		printf("Error creating CUDPPPlan\n");
		exit(-1);
	}
	// Should be cleaned up using `result = cudppDestroyPlan(planhandle);` 
	return planhandle;
}
*/
static void find_neighbours(int *d_neighbours, int n, float *d_distances,
		int eps) {
	static unsigned int *d_flags = NULL;
	const unsigned int blocksize = 16;
	unsigned int flags_bytes = n * n * sizeof(*d_flags);
	//static CUDPPHandle planhandle = 0;
	const dim3 threads(blocksize, blocksize);
	dim3 blocks(n / blocksize, n / blocksize);
	if (!d_flags) {
		size_t pitch;
		hipMallocPitch((void**) &d_flags, &pitch, n * sizeof(*d_flags), n);
		//planhandle = prepare_scan_plan(n, pitch / sizeof(*d_flags));
		//hipBindTexture(0, flags_texture, d_flags, n * n * sizeof(*d_flags));
		check_cuda_error();
	}
	hipMemset(d_flags, 0, flags_bytes);
	neighbourhood<<<blocks, threads>>>(d_neighbours, d_flags, d_distances, n,
			eps * eps);
	//compact<<<blocks, threads>>>(d_neighbours, d_flags, n);
}

__global__ static void count_distance(float *distance, const int n) {
   int ix = blockIdx.x * blockDim.x + threadIdx.x,
	   iy = blockIdx.y * blockDim.y + threadIdx.y,
	   iz = blockIdx.z * blockDim.z + threadIdx.z,
	   off_a = ix * sizeof(Boid), off_b = iy * sizeof(Boid);
   Boid b;//boid_from_texture(off_b);
   Boid a; //boid_from_texture(off_a);
   if (ix < n && iy < n)
	   distance[ix + n * iy] = Square(b.y - a.y) + Square(b.x - a.x);
}

static void reload_distance_cache(float *d_cache, int n) {
	const int blocksize = 16;
	dim3 threads(blocksize, blocksize);
	dim3 blocks(n / blocksize, n / blocksize);
	count_distance<<<blocks, threads>>>(d_cache, n);
}

__device__ static void separation(Boid *boids, int self, const int *neighbours,
		const int n, const float *distance_cache) {
	float x = 0, y = 0, z = 0;
	int count = 0, divisor, i;
	const int weight = 50;
	for (i = 0; neighbours[i] != INT_MAX; ++i) {
		int index = neighbours[i];
		float distance = sqrtf(distance_cache[self + n * index]) + 0.01f;
		x += (boids[self].x - boids[index].x) / distance;
		y += (boids[self].y - boids[index].y) / distance;
		z += (boids[self].z - boids[index].z) / distance;
		++count;
	}
	divisor = count * weight;
	boids[self].fx = x / divisor;
	boids[self].fy = y / divisor;
	boids[self].fz = z / divisor;
}

__device__ static void alignment(Boid *boids, Boid *self, const int *neighbours) {
	float vx = 0, vy = 0, vz = 0;
	int count = 0, i;
	const int weight = 10;
	for (i = 0; neighbours[i] != INT_MAX; ++i) {
		int index = neighbours[i];
		vx += boids[index].vX;
		vy += boids[index].vY;
		vz += boids[index].vZ;
		++count;
	}
	self->fx += vx / count / weight;
	self->fy += vy / count / weight;
	self->fz += vz / count / weight;
}

__device__ static void cohesion(const Boid *boids, Boid *self,
		const int *neighbours) {
	const int weight = 1000;
	float x = 0, y = 0;
	int i;
	for (i = 0; neighbours[i] != INT_MAX; ++i) {
		x += boids[neighbours[i]].x;
		y += boids[neighbours[i]].y;
	}
	x = x / i - self->x;
	y = y / i - self->y;
	self->fx += x / weight;
	self->fy += y / weight;
}

__global__ static void calculate_forces(Boid *boids,
		const float *distance_cache, const int n, const int *neighbours) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	if (ix < n && *(neighbours + n * ix) != INT_MAX) {
		separation(boids, ix, neighbours + n * ix, n, distance_cache);
		alignment(boids, boids + ix, neighbours + n * ix);
		cohesion(boids, boids + ix, neighbours + n * ix);
	}
}

static void calculate_all_forces(Boid* d_boids, int n, int eps,
		float *d_distance_cache) {
	const int blocksize = 64;
	int neighbours_bytes = n * n * sizeof(int);
	dim3 threads(blocksize), blocks(n / blocksize);
	static int *d_neighbours = NULL;
	if (!d_neighbours) {
		hipMalloc((void**) &d_neighbours, neighbours_bytes);
		assert(d_neighbours);
	}
	check_cuda_error();
	find_neighbours(d_neighbours, n, d_distance_cache, eps);
	check_cuda_error();
	calculate_forces<<<blocks, threads>>>(d_boids, d_distance_cache, n,
			d_neighbours);
	check_cuda_error();
}

__global__ static void attraction(Boid* boids, int n, float x, float y, float z) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	if (ix < n) {
		const float sin = sinf(TURNING_SPEED), cos = cosf(TURNING_SPEED), nSin = sinf(-TURNING_SPEED), nCos = cosf(-TURNING_SPEED);
		float next_x = boids[ix].x + boids[ix].vX, next_y = boids[ix].y + boids[ix].vY, next_z = boids[ix].z + boids[ix].vZ;
		float det = boids[ix].x * y * z + next_x * boids[ix].y *  + x * next_y
			- x * boids[ix].y - boids[ix].x * next_y - next_x * y;
		if (det > 0) {
			boids[ix].fx += boids[ix].vX - (boids[ix].vX * cos - boids[ix].vY * sin);
			boids[ix].fy += boids[ix].vY - (boids[ix].vX * sin + boids[ix].vY * cos);
			boids[ix].fz += boids[ix].vZ;
		} else if (det < 0) {
			boids[ix].fx += boids[ix].vX - (boids[ix].vX * nCos - boids[ix].vY * nSin);
			boids[ix].fy += boids[ix].vY - (boids[ix].vX * nSin + boids[ix].vY * nCos);
			boids[ix].fz += boids[ix].vZ;
		}
	}
}

static void calculate_attraction(Boid* d_boids, int n, int x, int y) {
	const int blocksize = 64;
	const int z = 0;
	dim3 threads(blocksize), blocks(n / blocksize);
	attraction<<<blocks, threads>>>(d_boids, n, x, y, z);
	check_cuda_error();
}

__device__ static void normalize_speed(Boid* boids) {
	const float SQUARE_LIMIT = Square(MAX_SPEED);
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	float speedSquared = Square(boids[ix].vX) + Square(boids[ix].vY) + Square(boids[ix].vZ);
	if (speedSquared > SQUARE_LIMIT) {
		float coeff = MAX_SPEED / sqrtf(speedSquared);
		boids[ix].vY *= coeff;
		boids[ix].vX *= coeff;
		boids[ix].vZ *= coeff;
	}
}

__global__ static void apply_forces(Boid* boids, float dt, int width, int height, int depth) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	boids[ix].vX += boids[ix].fx * dt;
	boids[ix].vY += boids[ix].fy * dt;
	boids[ix].vZ += boids[ix].fz * dt;
	boids[ix].fx = boids[ix].fy = 0;
	normalize_speed(boids);
	boids[ix].x += boids[ix].vX * dt;
	if (boids[ix].x >= width)
		boids[ix].x -= width;
	else if (boids[ix].x < 0)
		boids[ix].x += width;
	boids[ix].y += boids[ix].vY * dt;
	if (boids[ix].y >= height)
		boids[ix].y -= height;
	else if (boids[ix].y < 0)
		boids[ix].y += height;
	boids[ix].z += boids[ix].vZ * dt;
	if (boids[ix].z >= depth)
		boids[ix].z -= depth;
	else if (boids[ix].z < 0)
		boids[ix].z += depth;
}

static void apply_all_forces(Boid* pBoids, int n, float dt, int width, int height, int depth) {
	const int SIZE = 64;
	dim3 threads(SIZE), blocks(n / SIZE);
	apply_forces<<<blocks, threads>>>(pBoids, dt, width, height, depth);
	check_cuda_error();
}

static float* prepare_distance_cache(int n) {
	float *d_distance_cache = NULL;
	assert(n > 0);
	hipMalloc((void**) &d_distance_cache, n * n * sizeof(float));
	assert(d_distance_cache);
	return d_distance_cache;
}

static Boid * prepare_device_boids(int n) {
	int boidsBytes = n * sizeof(Boid);
	Boid *pBoids = NULL;
	assert(n > 0);
	hipMalloc((void**) &pBoids, boidsBytes);
	assert(pBoids);
	//hipBindTexture(0, boids_texture, pBoids, n * sizeof(Boid));
	check_cuda_error();
	return pBoids;
}

__global__ static void init_kernel(Boid *boids, int n) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	if (ix < n)
		boids[ix].x = boids[ix].y + (ix % 3);
}

static void CountIntensity(Sim *sp) {
	int i, j, max = 3;
	float coeff = 0xff;
	memset(sp->intensity, 0, sizeof(char) * sp->width * sp->height);
	for (i = 0; i < sp->n; ++i) {
		Boid b = sp->boids[i];
		int value = ++sp->intensity[((int) b.y) * sp->width + ((int) b.x)];
		if (value > max)
			max = value;
	}
	coeff /= max;
	#pragma omp parallel for private(j)
	for (i = 0; i < sp->height; ++i) {
		for (j = 0; j < sp->width; ++j)
			sp->intensity[i * sp->width + j] *= coeff;
	}
}

void simulate(Sim *sp) {
	static float *d_distance_cache = NULL;
	static Boid *pBoids = NULL;
	int boids_bytes = sp->n * sizeof(Boid);
	if (!d_distance_cache) {
		d_distance_cache = prepare_distance_cache(sp->n);
		pBoids = prepare_device_boids(sp->n);
		init_kernel<<<64, 64>>>(pBoids, sp->n);
		hipMemcpy(pBoids, sp->boids, boids_bytes, hipMemcpyHostToDevice);
		check_cuda_error();
	}
	reload_distance_cache(d_distance_cache, sp->n);
	calculate_all_forces(pBoids, sp->n, sp->eps, d_distance_cache);
	if (sp->attractors)
		calculate_attraction(pBoids, sp->n, sp->attractors->x,
				sp->attractors->y);
	apply_all_forces(pBoids, sp->n, sp->dt, sp->width, sp->height, sp->depth);
	hipDeviceSynchronize();
	check_cuda_error();
	hipMemcpy(sp->boids, pBoids, boids_bytes, hipMemcpyDeviceToHost);
	check_cuda_error();
	CountIntensity(sp);
}
